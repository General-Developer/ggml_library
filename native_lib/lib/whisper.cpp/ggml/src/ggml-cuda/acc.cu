#include "hip/hip_runtime.h"
/* <!-- START LICENSE -->


This Software / Program / Source Code Created By Developer From Company GLOBAL CORPORATION
Social Media:

   - Youtube: https://youtube.com/@Global_Corporation 
   - Github: https://github.com/globalcorporation
   - TELEGRAM: https://t.me/GLOBAL_CORP_ORG_BOT

All code script in here created 100% original without copy / steal from other code if we copy we add description source at from top code

If you wan't edit you must add credit me (don't change)

If this Software / Program / Source Code has you

Jika Program ini milik anda dari hasil beli jasa developer di (Global Corporation / apapun itu dari turunan itu jika ada kesalahan / bug / ingin update segera lapor ke sub)

Misal anda beli Beli source code di Slebew CORPORATION anda lapor dahulu di slebew jangan lapor di GLOBAL CORPORATION!

Jika ada kendala program ini (Pastikan sebelum deal project tidak ada negosiasi harga)
Karena jika ada negosiasi harga kemungkinan

1. Software Ada yang di kurangin
2. Informasi tidak lengkap
3. Bantuan Tidak Bisa remote / full time (Ada jeda)

Sebelum program ini sampai ke pembeli developer kami sudah melakukan testing

jadi sebelum nego kami sudah melakukan berbagai konsekuensi jika nego tidak sesuai ? 
Bukan maksud kami menipu itu karena harga yang sudah di kalkulasi + bantuan tiba tiba di potong akhirnya bantuan / software kadang tidak lengkap


<!-- END LICENSE --> */
#include "acc.cuh"

static __global__ void acc_f32(const float * x, const float * y, float * dst, const int ne,
    const int ne10, const int ne11, const int ne12,
    const int nb1, const int nb2, int offset) {
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= ne) {
        return;
    }
    int src1_idx = i - offset;
    int oz = src1_idx / nb2;
    int oy = (src1_idx - (oz * nb2)) / nb1;
    int ox = src1_idx % nb1;
    if (src1_idx >= 0 && ox < ne10 && oy < ne11 && oz < ne12) {
        dst[i] = x[i] + y[ox + oy * ne10 + oz * ne10 * ne11];
    } else {
        dst[i] = x[i];
    }
}

static void acc_f32_cuda(const float * x, const float * y, float * dst, const int n_elements,
    const int ne10, const int ne11, const int ne12,
    const int nb1, const int nb2, const int offset, hipStream_t stream) {
    int num_blocks = (n_elements + CUDA_ACC_BLOCK_SIZE - 1) / CUDA_ACC_BLOCK_SIZE;
    acc_f32<<<num_blocks, CUDA_ACC_BLOCK_SIZE, 0, stream>>>(x, y, dst, n_elements, ne10, ne11, ne12, nb1, nb2, offset);
}

void ggml_cuda_op_acc(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    const float * src0_d = (const float *)src0->data;
    const float * src1_d = (const float *)src1->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->ne[3] == 1); // just 3D tensors supported

    int nb1 = dst->op_params[0] / 4; // 4 bytes of float32
    int nb2 = dst->op_params[1] / 4; // 4 bytes of float32
    // int nb3 = dst->op_params[2] / 4; // 4 bytes of float32 - unused
    int offset = dst->op_params[3] / 4; // offset in bytes

    acc_f32_cuda(src0_d, src1_d, dst_d, ggml_nelements(dst), src1->ne[0], src1->ne[1], src1->ne[2], nb1, nb2, offset, stream);
}

/* <!-- START LICENSE -->


This Software / Program / Source Code Created By Developer From Company GLOBAL CORPORATION
Social Media:

   - Youtube: https://youtube.com/@Global_Corporation 
   - Github: https://github.com/globalcorporation
   - TELEGRAM: https://t.me/GLOBAL_CORP_ORG_BOT

All code script in here created 100% original without copy / steal from other code if we copy we add description source at from top code

If you wan't edit you must add credit me (don't change)

If this Software / Program / Source Code has you

Jika Program ini milik anda dari hasil beli jasa developer di (Global Corporation / apapun itu dari turunan itu jika ada kesalahan / bug / ingin update segera lapor ke sub)

Misal anda beli Beli source code di Slebew CORPORATION anda lapor dahulu di slebew jangan lapor di GLOBAL CORPORATION!

Jika ada kendala program ini (Pastikan sebelum deal project tidak ada negosiasi harga)
Karena jika ada negosiasi harga kemungkinan

1. Software Ada yang di kurangin
2. Informasi tidak lengkap
3. Bantuan Tidak Bisa remote / full time (Ada jeda)

Sebelum program ini sampai ke pembeli developer kami sudah melakukan testing

jadi sebelum nego kami sudah melakukan berbagai konsekuensi jika nego tidak sesuai ? 
Bukan maksud kami menipu itu karena harga yang sudah di kalkulasi + bantuan tiba tiba di potong akhirnya bantuan / software kadang tidak lengkap


<!-- END LICENSE --> */
#include "out-prod.cuh"

#include <cstdint>

void ggml_cuda_out_prod(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    GGML_TENSOR_BINARY_OP_LOCALS

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type  == GGML_TYPE_F32);

    GGML_ASSERT(ne01 == ne11);
    GGML_ASSERT(ne0 == ne00);
    GGML_ASSERT(ne1 == ne10);

    GGML_ASSERT(ne2 % src0->ne[2] == 0);
    GGML_ASSERT(ne3 % src0->ne[3] == 0);

    GGML_ASSERT(ne2 == src1->ne[2]);
    GGML_ASSERT(ne3 == src1->ne[3]);

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    float       *  dst_d = (float       *)  dst->data;

    hipStream_t   stream = ctx.stream();
    hipblasHandle_t handle = ctx.cublas_handle();

    const float alpha = 1.0f;
    const float beta = 0.0f;

    CUBLAS_CHECK(hipblasSetStream(handle, stream));

    const int64_t lda = nb01 / sizeof(float);
    const int64_t ldc = nb1  / sizeof(float);

    const bool src1_T = ggml_is_transposed(src1);
    const hipblasOperation_t src1_cublas_op =  src1_T ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    const int64_t           ldb            = (src1_T ?        nb10 :        nb11) /  sizeof(float);
    GGML_ASSERT(                             (src1_T ?        nb11 :        nb10) == sizeof(float));

    // data strides in dimensions 2/3
    const size_t s02 = nb02 / sizeof(float);
    const size_t s03 = nb03 / sizeof(float);
    const size_t s12 = nb12 / sizeof(float);
    const size_t s13 = nb13 / sizeof(float);
    const size_t s2  = nb2  / sizeof(float);
    const size_t s3  = nb3  / sizeof(float);

    // dps == dst per src0, used for group query attention
    const int64_t dps2 = ne2 / ne02;
    const int64_t dps3 = ne3 / ne03;

    // TODO batched matrix multiplication
    for (int64_t i3 = 0; i3 < ne3; ++i3) {
        for (int64_t i2 = 0; i2 < ne2; ++i2) {
            CUBLAS_CHECK(
                hipblasSgemm(handle, HIPBLAS_OP_N, src1_cublas_op,
                        ne0, ne1, ne01,
                        &alpha, src0_d + (i3/dps3)*s03 + (i2/dps2)*s02, lda,
                                src1_d +  i3      *s13 +  i2      *s12, ldb,
                        &beta,  dst_d  +  i3      *s3  +  i2      *s2,  ldc));
        }
    }
}

#include "hip/hip_runtime.h"
/* <!-- START LICENSE -->


This Software / Program / Source Code Created By Developer From Company GLOBAL CORPORATION
Social Media:

   - Youtube: https://youtube.com/@Global_Corporation 
   - Github: https://github.com/globalcorporation
   - TELEGRAM: https://t.me/GLOBAL_CORP_ORG_BOT

All code script in here created 100% original without copy / steal from other code if we copy we add description source at from top code

If you wan't edit you must add credit me (don't change)

If this Software / Program / Source Code has you

Jika Program ini milik anda dari hasil beli jasa developer di (Global Corporation / apapun itu dari turunan itu jika ada kesalahan / bug / ingin update segera lapor ke sub)

Misal anda beli Beli source code di Slebew CORPORATION anda lapor dahulu di slebew jangan lapor di GLOBAL CORPORATION!

Jika ada kendala program ini (Pastikan sebelum deal project tidak ada negosiasi harga)
Karena jika ada negosiasi harga kemungkinan

1. Software Ada yang di kurangin
2. Informasi tidak lengkap
3. Bantuan Tidak Bisa remote / full time (Ada jeda)

Sebelum program ini sampai ke pembeli developer kami sudah melakukan testing

jadi sebelum nego kami sudah melakukan berbagai konsekuensi jika nego tidak sesuai ? 
Bukan maksud kami menipu itu karena harga yang sudah di kalkulasi + bantuan tiba tiba di potong akhirnya bantuan / software kadang tidak lengkap


<!-- END LICENSE --> */
#include "pad.cuh"

static __global__ void pad_f32(const float * x, float * dst, const int ne0, const int ne00, const int ne01, const int ne02, const int ne03) {
    // blockIdx.z: idx of ne2*ne3, aka ne02*ne03
    // blockIdx.y: idx of ne1
    // blockIDx.x: idx of ne0 / BLOCK_SIZE
    int nidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (nidx >= ne0) {
        return;
    }

    // operation
    int offset_dst =
        nidx +
        blockIdx.y * ne0 +
        blockIdx.z * ne0 * gridDim.y;
    if (nidx < ne00 && blockIdx.y < ne01 && blockIdx.z < ne02*ne03) {
        int offset_src =
            nidx +
            blockIdx.y * ne00 +
            blockIdx.z * ne00 * ne01;
        dst[offset_dst] = x[offset_src];
    } else {
        dst[offset_dst] = 0.0f;
    }
}

static void pad_f32_cuda(const float * x, float * dst,
    const int ne00, const int ne01, const int ne02, const int ne03,
    const int ne0, const int ne1, const int ne2, const int ne3, hipStream_t stream) {
    int num_blocks = (ne0 + CUDA_PAD_BLOCK_SIZE - 1) / CUDA_PAD_BLOCK_SIZE;
    dim3 gridDim(num_blocks, ne1, ne2*ne3);
    pad_f32<<<gridDim, CUDA_PAD_BLOCK_SIZE, 0, stream>>>(x, dst, ne0, ne00, ne01, ne02, ne03);
}

void ggml_cuda_op_pad(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);
    GGML_ASSERT(src0->ne[3] == 1 && dst->ne[3] == 1); // just 3D tensors

    pad_f32_cuda(src0_d, dst_d,
        src0->ne[0], src0->ne[1], src0->ne[2], src0->ne[3],
        dst->ne[0], dst->ne[1], dst->ne[2], dst->ne[3], stream);
}

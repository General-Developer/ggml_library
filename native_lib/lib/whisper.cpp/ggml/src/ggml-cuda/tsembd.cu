#include "hip/hip_runtime.h"
/* <!-- START LICENSE -->


This Software / Program / Source Code Created By Developer From Company GLOBAL CORPORATION
Social Media:

   - Youtube: https://youtube.com/@Global_Corporation 
   - Github: https://github.com/globalcorporation
   - TELEGRAM: https://t.me/GLOBAL_CORP_ORG_BOT

All code script in here created 100% original without copy / steal from other code if we copy we add description source at from top code

If you wan't edit you must add credit me (don't change)

If this Software / Program / Source Code has you

Jika Program ini milik anda dari hasil beli jasa developer di (Global Corporation / apapun itu dari turunan itu jika ada kesalahan / bug / ingin update segera lapor ke sub)

Misal anda beli Beli source code di Slebew CORPORATION anda lapor dahulu di slebew jangan lapor di GLOBAL CORPORATION!

Jika ada kendala program ini (Pastikan sebelum deal project tidak ada negosiasi harga)
Karena jika ada negosiasi harga kemungkinan

1. Software Ada yang di kurangin
2. Informasi tidak lengkap
3. Bantuan Tidak Bisa remote / full time (Ada jeda)

Sebelum program ini sampai ke pembeli developer kami sudah melakukan testing

jadi sebelum nego kami sudah melakukan berbagai konsekuensi jika nego tidak sesuai ? 
Bukan maksud kami menipu itu karena harga yang sudah di kalkulasi + bantuan tiba tiba di potong akhirnya bantuan / software kadang tidak lengkap


<!-- END LICENSE --> */
#include "tsembd.cuh"

static __global__ void timestep_embedding_f32(const float * timesteps, float * dst, const int nb1, const int dim, const int max_period) {
    // blockIDx.y: idx of timesteps->ne[0]
    // blockIDx.x: idx of ((dim + 1) / 2) / BLOCK_SIZE
    int i = blockIdx.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    float * embed_data = (float *)((char *)dst +  i*nb1);

    if (dim % 2 != 0 && j == ((dim + 1) / 2)) {
        embed_data[dim] = 0.f;
    }

    int half = dim / 2;
    if (j >= half) {
        return;
    }

    float timestep = timesteps[i];
    float freq = (float)expf(-logf(max_period) * j / half);
    float arg = timestep * freq;
    embed_data[j] = cosf(arg);
    embed_data[j + half] = sinf(arg);
}

static void timestep_embedding_f32_cuda(const float * x, float * dst, const int ne00, const int nb1,
                                        const int dim, const int max_period, hipStream_t stream) {
    int half_ceil = (dim + 1) / 2;
    int num_blocks = (half_ceil + CUDA_TIMESTEP_EMBEDDING_BLOCK_SIZE - 1) / CUDA_TIMESTEP_EMBEDDING_BLOCK_SIZE;
    dim3 gridDim(num_blocks, ne00, 1);
    timestep_embedding_f32<<<gridDim, CUDA_TIMESTEP_EMBEDDING_BLOCK_SIZE, 0, stream>>>(x, dst, nb1, dim, max_period);
}

void ggml_cuda_op_timestep_embedding(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);

    const int dim = dst->op_params[0];
    const int max_period = dst->op_params[1];

    timestep_embedding_f32_cuda(src0_d, dst_d, src0->ne[0], dst->nb[1], dim, max_period, stream);
}

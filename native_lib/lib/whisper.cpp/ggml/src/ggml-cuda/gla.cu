#include "hip/hip_runtime.h"
/* <!-- START LICENSE -->


This Software / Program / Source Code Created By Developer From Company GLOBAL CORPORATION
Social Media:

   - Youtube: https://youtube.com/@Global_Corporation 
   - Github: https://github.com/globalcorporation
   - TELEGRAM: https://t.me/GLOBAL_CORP_ORG_BOT

All code script in here created 100% original without copy / steal from other code if we copy we add description source at from top code

If you wan't edit you must add credit me (don't change)

If this Software / Program / Source Code has you

Jika Program ini milik anda dari hasil beli jasa developer di (Global Corporation / apapun itu dari turunan itu jika ada kesalahan / bug / ingin update segera lapor ke sub)

Misal anda beli Beli source code di Slebew CORPORATION anda lapor dahulu di slebew jangan lapor di GLOBAL CORPORATION!

Jika ada kendala program ini (Pastikan sebelum deal project tidak ada negosiasi harga)
Karena jika ada negosiasi harga kemungkinan

1. Software Ada yang di kurangin
2. Informasi tidak lengkap
3. Bantuan Tidak Bisa remote / full time (Ada jeda)

Sebelum program ini sampai ke pembeli developer kami sudah melakukan testing

jadi sebelum nego kami sudah melakukan berbagai konsekuensi jika nego tidak sesuai ? 
Bukan maksud kami menipu itu karena harga yang sudah di kalkulasi + bantuan tiba tiba di potong akhirnya bantuan / software kadang tidak lengkap


<!-- END LICENSE --> */
#include "common.cuh"
#include "gla.cuh"

template<int HEAD_SIZE>
static __global__ void gated_linear_attn_f32(const int B, const int T, const int C, const int H, const float scale,
     const float * k, const float * v, const float * r, const float * td, const float * s, float * dst) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    const int head_size = HEAD_SIZE;
    const int batch_i = bid / H;
    const int head_i = bid % H;
    const int state_size = C * head_size;
    const int n_seq_tokens = T / B;

    float state[head_size];
    __shared__ float _k[head_size], _r[head_size], _td[head_size];

    #pragma unroll
    for (int i = 0; i < head_size; i++) {
        state[i] = s[batch_i * state_size + head_i * head_size * head_size + i * head_size + tid];
    }

    for (int t = batch_i * n_seq_tokens * C + head_i * head_size + tid; t < (batch_i + 1) * n_seq_tokens * C + head_i * head_size + tid; t += C) {
        __syncthreads();
        _k[tid] = k[t];
        _r[tid] = r[t];
        _td[tid] = td[t];
        __syncthreads();

        const float _v = v[t];
        float y = 0;
        for (int j = 0; j < head_size; j += 4) {
            const float4 & k = (float4 &)(_k[j]);
            const float4 & r = (float4 &)(_r[j]);
            const float4 & td = (float4 &)(_td[j]);
            float4 & s = (float4 &)(state[j]);
            float4 kv;

            kv.x = k.x * _v;
            kv.y = k.y * _v;
            kv.z = k.z * _v;
            kv.w = k.w * _v;

            s.x = s.x * td.x + kv.x;
            s.y = s.y * td.y + kv.y;
            s.z = s.z * td.z + kv.z;
            s.w = s.w * td.w + kv.w;

            y += r.x * s.x;
            y += r.y * s.y;
            y += r.z * s.z;
            y += r.w * s.w;
        }
        dst[t] = y * scale;
    }

    #pragma unroll
    for (int i = 0; i < head_size; i++) {
        dst[T * C + batch_i * state_size + head_i * head_size * head_size + i * head_size + tid] = state[i];
    }
}

void ggml_cuda_op_gated_linear_attn(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const float * k_d  = (const float *)dst->src[0]->data;
    const float * v_d  = (const float *)dst->src[1]->data;
    const float * r_d  = (const float *)dst->src[2]->data;
    const float * td_d = (const float *)dst->src[3]->data;
    const float * s_d  = (const float *)dst->src[4]->data;

    const int64_t B = dst->src[4]->ne[1];
    const int64_t T = dst->src[0]->ne[2];
    const int64_t C = dst->ne[0];
    const int64_t H = dst->src[0]->ne[1];

    float scale;
    memcpy(&scale, (float*)dst->op_params, sizeof(float));

    float * dst_d = (float *)dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(dst->src[4]->type == GGML_TYPE_F32);
    GGML_ASSERT(C % H == 0);
    GGML_ASSERT(C / H == 64 || C / H == 128);


    if (C / H == 64) {
        gated_linear_attn_f32<64><<<B * H, C / H, 0, stream>>>(B, T, C, H, scale, k_d, v_d, r_d, td_d, s_d, dst_d);
    } else {
        gated_linear_attn_f32<128><<<B * H, C / H, 0, stream>>>(B, T, C, H, scale, k_d, v_d, r_d, td_d, s_d, dst_d);
    }
}

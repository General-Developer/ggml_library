#include "hip/hip_runtime.h"
/* <!-- START LICENSE -->


This Software / Program / Source Code Created By Developer From Company GLOBAL CORPORATION
Social Media:

   - Youtube: https://youtube.com/@Global_Corporation 
   - Github: https://github.com/globalcorporation
   - TELEGRAM: https://t.me/GLOBAL_CORP_ORG_BOT

All code script in here created 100% original without copy / steal from other code if we copy we add description source at from top code

If you wan't edit you must add credit me (don't change)

If this Software / Program / Source Code has you

Jika Program ini milik anda dari hasil beli jasa developer di (Global Corporation / apapun itu dari turunan itu jika ada kesalahan / bug / ingin update segera lapor ke sub)

Misal anda beli Beli source code di Slebew CORPORATION anda lapor dahulu di slebew jangan lapor di GLOBAL CORPORATION!

Jika ada kendala program ini (Pastikan sebelum deal project tidak ada negosiasi harga)
Karena jika ada negosiasi harga kemungkinan

1. Software Ada yang di kurangin
2. Informasi tidak lengkap
3. Bantuan Tidak Bisa remote / full time (Ada jeda)

Sebelum program ini sampai ke pembeli developer kami sudah melakukan testing

jadi sebelum nego kami sudah melakukan berbagai konsekuensi jika nego tidak sesuai ? 
Bukan maksud kami menipu itu karena harga yang sudah di kalkulasi + bantuan tiba tiba di potong akhirnya bantuan / software kadang tidak lengkap


<!-- END LICENSE --> */
#include "diagmask.cuh"

static __global__ void diag_mask_inf_f32(const float * x, float * dst, const int ncols, const int rows_per_channel, const int n_past) {
    const int col = blockDim.y*blockIdx.y + threadIdx.y;
    const int row = blockDim.x*blockIdx.x + threadIdx.x;

    if (col >= ncols) {
        return;
    }

    const int i = row*ncols + col;
    //dst[i] = col > (n_past + row % rows_per_channel) ? -INFINITY : x[i];
    //dst[i] = x[i] - (col > n_past + row % rows_per_channel) * INT_MAX; // equivalent within rounding error but slightly faster on GPU
    dst[i] = x[i] - (col > n_past + row % rows_per_channel) * FLT_MAX;
}

static void diag_mask_inf_f32_cuda(const float * x, float * dst, const int ncols_x, const int nrows_x, const int rows_per_channel, const int n_past, hipStream_t stream) {
    const dim3 block_dims(1, CUDA_DIAG_MASK_INF_BLOCK_SIZE, 1);
    const int block_num_x = (ncols_x + CUDA_DIAG_MASK_INF_BLOCK_SIZE - 1) / CUDA_DIAG_MASK_INF_BLOCK_SIZE;
    const dim3 block_nums(nrows_x, block_num_x, 1);
    diag_mask_inf_f32<<<block_nums, block_dims, 0, stream>>>(x, dst, ncols_x, rows_per_channel, n_past);
}

void ggml_cuda_op_diag_mask_inf(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int nrows0 = ggml_nrows(src0);

    const int n_past = ((int32_t *) dst->op_params)[0];

    diag_mask_inf_f32_cuda(src0_d, dst_d, ne00, nrows0, ne01, n_past, stream);
}

#include "hip/hip_runtime.h"
/* <!-- START LICENSE -->


This Software / Program / Source Code Created By Developer From Company GLOBAL CORPORATION
Social Media:

   - Youtube: https://youtube.com/@Global_Corporation 
   - Github: https://github.com/globalcorporation
   - TELEGRAM: https://t.me/GLOBAL_CORP_ORG_BOT

All code script in here created 100% original without copy / steal from other code if we copy we add description source at from top code

If you wan't edit you must add credit me (don't change)

If this Software / Program / Source Code has you

Jika Program ini milik anda dari hasil beli jasa developer di (Global Corporation / apapun itu dari turunan itu jika ada kesalahan / bug / ingin update segera lapor ke sub)

Misal anda beli Beli source code di Slebew CORPORATION anda lapor dahulu di slebew jangan lapor di GLOBAL CORPORATION!

Jika ada kendala program ini (Pastikan sebelum deal project tidak ada negosiasi harga)
Karena jika ada negosiasi harga kemungkinan

1. Software Ada yang di kurangin
2. Informasi tidak lengkap
3. Bantuan Tidak Bisa remote / full time (Ada jeda)

Sebelum program ini sampai ke pembeli developer kami sudah melakukan testing

jadi sebelum nego kami sudah melakukan berbagai konsekuensi jika nego tidak sesuai ? 
Bukan maksud kami menipu itu karena harga yang sudah di kalkulasi + bantuan tiba tiba di potong akhirnya bantuan / software kadang tidak lengkap


<!-- END LICENSE --> */
#include "arange.cuh"

static __global__ void arange_f32(float * dst, const int ne0, const float start, const float step) {
    // blockIDx.x: idx of ne0 / BLOCK_SIZE
    int nidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (nidx >= ne0) {
        return;
    }
    dst[nidx] = start + step * nidx;
}

static void arange_f32_cuda(float * dst, const int ne0, const float start, const float step, hipStream_t stream) {
    int num_blocks = (ne0 + CUDA_ARANGE_BLOCK_SIZE - 1) / CUDA_ARANGE_BLOCK_SIZE;
    arange_f32<<<num_blocks, CUDA_ARANGE_BLOCK_SIZE, 0, stream>>>(dst, ne0, start,  step);
}

void ggml_cuda_op_arange(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(dst->type == GGML_TYPE_F32);

    float start;
    float stop;
    float step;
    memcpy(&start, (float *)dst->op_params + 0, sizeof(float));
    memcpy(&stop,  (float *)dst->op_params + 1, sizeof(float));
    memcpy(&step,  (float *)dst->op_params + 2, sizeof(float));

    int64_t steps = (int64_t)ceil((stop - start) / step);
    GGML_ASSERT(ggml_nelements(dst) == steps);

    arange_f32_cuda(dst_d, dst->ne[0], start, step, stream);
}

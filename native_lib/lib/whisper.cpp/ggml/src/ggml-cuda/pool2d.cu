#include "hip/hip_runtime.h"
/* <!-- START LICENSE -->


This Software / Program / Source Code Created By Developer From Company GLOBAL CORPORATION
Social Media:

   - Youtube: https://youtube.com/@Global_Corporation 
   - Github: https://github.com/globalcorporation
   - TELEGRAM: https://t.me/GLOBAL_CORP_ORG_BOT

All code script in here created 100% original without copy / steal from other code if we copy we add description source at from top code

If you wan't edit you must add credit me (don't change)

If this Software / Program / Source Code has you

Jika Program ini milik anda dari hasil beli jasa developer di (Global Corporation / apapun itu dari turunan itu jika ada kesalahan / bug / ingin update segera lapor ke sub)

Misal anda beli Beli source code di Slebew CORPORATION anda lapor dahulu di slebew jangan lapor di GLOBAL CORPORATION!

Jika ada kendala program ini (Pastikan sebelum deal project tidak ada negosiasi harga)
Karena jika ada negosiasi harga kemungkinan

1. Software Ada yang di kurangin
2. Informasi tidak lengkap
3. Bantuan Tidak Bisa remote / full time (Ada jeda)

Sebelum program ini sampai ke pembeli developer kami sudah melakukan testing

jadi sebelum nego kami sudah melakukan berbagai konsekuensi jika nego tidak sesuai ? 
Bukan maksud kami menipu itu karena harga yang sudah di kalkulasi + bantuan tiba tiba di potong akhirnya bantuan / software kadang tidak lengkap


<!-- END LICENSE --> */
#include "pool2d.cuh"

template <typename Ti, typename To>
static  __global__ void pool2d_nchw_kernel(
        const int ih, const int iw, const int oh, const int ow,
        const int kh, const int kw, const int sh, const int sw,
        const int ph, const int pw, const int parallel_elements,
        const Ti* src, To* dst, const enum ggml_op_pool op) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= parallel_elements) {
        return;
    }

    const int I_HW = ih * iw;
    const int O_HW = oh * ow;
    const int nc = idx / O_HW;
    const int cur_oh = idx % O_HW / ow;
    const int cur_ow = idx % O_HW % ow;
    const Ti* i_ptr = src + nc * I_HW;
    To* o_ptr = dst + nc * O_HW;
    const int start_h = cur_oh * sh - ph;
    const int bh = max(0, start_h);
    const int eh = min(ih, start_h + kh);
    const int start_w = cur_ow * sw - pw;
    const int bw = max(0, start_w);
    const int ew = min(iw, start_w + kw);
    const To scale = 1. / (kh * kw);
    To res = 0;

    switch (op) {
        case GGML_OP_POOL_AVG: res = 0; break;
        case GGML_OP_POOL_MAX: res = -FLT_MAX; break;
        default: assert(false);
    }

    for (int i = bh; i < eh; i += 1) {
        for (int j = bw; j < ew; j += 1) {
#if __CUDA_ARCH__ >= 350
            Ti cur = __ldg(i_ptr + i * iw + j);
#else
            Ti cur = i_ptr[i * iw + j];
#endif
            switch (op) {
                case GGML_OP_POOL_AVG: res += cur * scale; break;
                case GGML_OP_POOL_MAX: res = max(res, (To)cur); break;
                default: assert(false);
            }
        }
    }
    o_ptr[cur_oh * ow + cur_ow] = res;
}

static void pool2d_nchw_kernel_f32_f32_cuda(
        const int ih, const int iw, const int oh, const int ow,
        const int kh, const int kw, const int sh, const int sw,
        const int ph, const int pw, const int parallel_elements,
        const float * src, float * dst, const enum ggml_op_pool op,
        hipStream_t stream) {

    const int num_blocks = (parallel_elements + CUDA_POOL2D_BLOCK_SIZE - 1) / CUDA_POOL2D_BLOCK_SIZE;
    dim3 block_nums(num_blocks);
    pool2d_nchw_kernel<<<block_nums, CUDA_POOL2D_BLOCK_SIZE, 0, stream>>>(ih, iw, oh, ow, kh, kw, sh, sw, ph, pw, parallel_elements, src, dst, op);
}

void ggml_cuda_op_pool2d(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    const int32_t * opts = (const int32_t *)dst->op_params;
    enum ggml_op_pool op = static_cast<ggml_op_pool>(opts[0]);
    const int k0 = opts[1];
    const int k1 = opts[2];
    const int s0 = opts[3];
    const int s1 = opts[4];
    const int p0 = opts[5];
    const int p1 = opts[6];

    const int64_t IH = src0->ne[1];
    const int64_t IW = src0->ne[0];

    const int64_t N = dst->ne[3];
    const int64_t OC = dst->ne[2];
    const int64_t OH = dst->ne[1];
    const int64_t OW = dst->ne[0];

    const int parallel_elements = N * OC * OH * OW;

    pool2d_nchw_kernel_f32_f32_cuda(IH, IW, OH, OW, k1, k0, s1, s0, p1, p0, parallel_elements, src0_d, dst_d, op, stream);
}

#include "hip/hip_runtime.h"
/* <!-- START LICENSE -->


This Software / Program / Source Code Created By Developer From Company GLOBAL CORPORATION
Social Media:

   - Youtube: https://youtube.com/@Global_Corporation 
   - Github: https://github.com/globalcorporation
   - TELEGRAM: https://t.me/GLOBAL_CORP_ORG_BOT

All code script in here created 100% original without copy / steal from other code if we copy we add description source at from top code

If you wan't edit you must add credit me (don't change)

If this Software / Program / Source Code has you

Jika Program ini milik anda dari hasil beli jasa developer di (Global Corporation / apapun itu dari turunan itu jika ada kesalahan / bug / ingin update segera lapor ke sub)

Misal anda beli Beli source code di Slebew CORPORATION anda lapor dahulu di slebew jangan lapor di GLOBAL CORPORATION!

Jika ada kendala program ini (Pastikan sebelum deal project tidak ada negosiasi harga)
Karena jika ada negosiasi harga kemungkinan

1. Software Ada yang di kurangin
2. Informasi tidak lengkap
3. Bantuan Tidak Bisa remote / full time (Ada jeda)

Sebelum program ini sampai ke pembeli developer kami sudah melakukan testing

jadi sebelum nego kami sudah melakukan berbagai konsekuensi jika nego tidak sesuai ? 
Bukan maksud kami menipu itu karena harga yang sudah di kalkulasi + bantuan tiba tiba di potong akhirnya bantuan / software kadang tidak lengkap


<!-- END LICENSE --> */
#include "conv-transpose-1d.cuh"

static  __global__ void conv_transpose_1d_kernel(
        const int s0, const int p0, const int d0, const int output_size,
        const int src0_ne0, const int src0_ne1, const int src0_ne2, const int src0_ne3,
        const int src1_ne0, const int src1_ne1, const int src1_ne2, const int src1_ne3,
        const int dst_ne0, const int dst_ne1, const int dst_ne2, const int dst_ne3,
        const float * src0, const float * src1,  float * dst) {
    int global_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (global_index >= output_size) {
        return;
    }

    int out_index = global_index / dst_ne0;

    float accumulator = 0;

    for (int c = 0; c < src0_ne2; c++) {
        int idx = global_index % dst_ne0;

        int kernel_offset = (src0_ne0 * src0_ne1 * c) + (out_index * src0_ne0);
        int input_offset = src1_ne0 * c;

        for (int i = 0; i < src1_ne0; i++) {
            if (!(idx >= i*s0 && idx < i*s0 + src0_ne0)) {
                continue;
            }
            int weight_idx = idx - i*s0;

            float kernel_weight = src0[kernel_offset + weight_idx];
            float input_value =  src1[input_offset+i];

            accumulator += kernel_weight * input_value;
        }
    }
    dst[global_index] = accumulator;
}

static void conv_transpose_1d_f32_f32_cuda(
        const int s0, const int p0, const int d0, const int output_size,
        const int src0_ne0, const int src0_ne1, const int src0_ne2, const int src0_ne3,
        const int src1_ne0, const int src1_ne1, const int src1_ne2, const int src1_ne3,
        const int dst_ne0, const int dst_ne1, const int dst_ne2, const int dst_ne3,
        const float * src0, const float * src1,  float * dst,
        hipStream_t stream) {

    const int num_blocks = (output_size + CUDA_CONV_TRANPOSE_1D_BLOCK_SIZE - 1) / CUDA_CONV_TRANPOSE_1D_BLOCK_SIZE;
    conv_transpose_1d_kernel<<<num_blocks,CUDA_CONV_TRANPOSE_1D_BLOCK_SIZE, 0, stream>>>(
        s0,p0,d0,output_size,
        src0_ne0, src0_ne1,  src0_ne2, src0_ne3,
        src1_ne0, src1_ne1,  src1_ne2, src1_ne3,
        dst_ne0,  dst_ne1,   dst_ne2,  dst_ne3,
        src0,src1, dst);
}

void ggml_cuda_op_conv_transpose_1d(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;

    const ggml_tensor * src1 = dst->src[1];
    const float * src1_d = (const float *)src1->data;

    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(src1));

    const int32_t * opts = (const int32_t *)dst->op_params;

    const int s0 = opts[0];
    const int p0 = 0;//opts[3];
    const int d0 = 1;//opts[4];

    const int64_t kernel_size = ggml_nelements(src0);
    const int64_t input_size = ggml_nelements(src1);
    const int64_t output_size = ggml_nelements(dst);

    conv_transpose_1d_f32_f32_cuda(s0, p0, d0, output_size,
        src0->ne[0], src0->ne[1], src0->ne[2], src0->ne[3],
        src1->ne[0], src1->ne[1], src1->ne[2], src1->ne[3],
        dst->ne[0], dst->ne[1], dst->ne[2], dst->ne[3],
        src0_d, src1_d, dst_d, stream);
}

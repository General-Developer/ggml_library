/* <!-- START LICENSE -->


This Software / Program / Source Code Created By Developer From Company GLOBAL CORPORATION
Social Media:

   - Youtube: https://youtube.com/@Global_Corporation 
   - Github: https://github.com/globalcorporation
   - TELEGRAM: https://t.me/GLOBAL_CORP_ORG_BOT

All code script in here created 100% original without copy / steal from other code if we copy we add description source at from top code

If you wan't edit you must add credit me (don't change)

If this Software / Program / Source Code has you

Jika Program ini milik anda dari hasil beli jasa developer di (Global Corporation / apapun itu dari turunan itu jika ada kesalahan / bug / ingin update segera lapor ke sub)

Misal anda beli Beli source code di Slebew CORPORATION anda lapor dahulu di slebew jangan lapor di GLOBAL CORPORATION!

Jika ada kendala program ini (Pastikan sebelum deal project tidak ada negosiasi harga)
Karena jika ada negosiasi harga kemungkinan

1. Software Ada yang di kurangin
2. Informasi tidak lengkap
3. Bantuan Tidak Bisa remote / full time (Ada jeda)

Sebelum program ini sampai ke pembeli developer kami sudah melakukan testing

jadi sebelum nego kami sudah melakukan berbagai konsekuensi jika nego tidak sesuai ? 
Bukan maksud kami menipu itu karena harga yang sudah di kalkulasi + bantuan tiba tiba di potong akhirnya bantuan / software kadang tidak lengkap


<!-- END LICENSE --> */
#if !defined(GGML_USE_HIP) && !defined(GGML_USE_MUSA) && CUDART_VERSION >= 11700
#define USE_CUB
#endif // !defined(GGML_USE_HIP) && !defined(GGML_USE_MUSA) && CUDART_VERSION >= 11700

#ifdef USE_CUB
#include <hipcub/hipcub.hpp>
using namespace hipcub;
#endif // USE_CUB

#include "sumrows.cuh"
#include "sum.cuh"

#include <cstdint>

void sum_f32_cuda(ggml_cuda_pool & pool, const float * x, float * dst, const int64_t ne, hipStream_t stream) {
#ifdef USE_CUB
    size_t tmp_size = 0;
    DeviceReduce::Sum(nullptr,       tmp_size, x, dst, ne, stream);
    ggml_cuda_pool_alloc<uint8_t> tmp_alloc(pool, tmp_size);
    DeviceReduce::Sum(tmp_alloc.ptr, tmp_size, x, dst, ne, stream);
#else
    // Use (inefficient) sum_rows implementation as a fallback.
    // For AMD there is rocPRIM which could be used as a drop-in replacement via hipcub but this would require C++11 -> C++14.
    sum_rows_f32_cuda(x, dst, ne, 1, stream);
    GGML_UNUSED(pool);
#endif // USE_CUB
}

void ggml_cuda_op_sum(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);
    GGML_ASSERT(ggml_is_contiguous(src0));

    const float * src0_d = (const float *) src0->data;
    float * dst_d = (float *) dst->data;

    const int64_t ne = ggml_nelements(src0);

    ggml_cuda_pool & pool = ctx.pool();
    hipStream_t stream = ctx.stream();

    sum_f32_cuda(pool, src0_d, dst_d, ne, stream);
}

#include "hip/hip_runtime.h"
/* <!-- START LICENSE -->


This Software / Program / Source Code Created By Developer From Company GLOBAL CORPORATION
Social Media:

   - Youtube: https://youtube.com/@Global_Corporation 
   - Github: https://github.com/globalcorporation
   - TELEGRAM: https://t.me/GLOBAL_CORP_ORG_BOT

All code script in here created 100% original without copy / steal from other code if we copy we add description source at from top code

If you wan't edit you must add credit me (don't change)

If this Software / Program / Source Code has you

Jika Program ini milik anda dari hasil beli jasa developer di (Global Corporation / apapun itu dari turunan itu jika ada kesalahan / bug / ingin update segera lapor ke sub)

Misal anda beli Beli source code di Slebew CORPORATION anda lapor dahulu di slebew jangan lapor di GLOBAL CORPORATION!

Jika ada kendala program ini (Pastikan sebelum deal project tidak ada negosiasi harga)
Karena jika ada negosiasi harga kemungkinan

1. Software Ada yang di kurangin
2. Informasi tidak lengkap
3. Bantuan Tidak Bisa remote / full time (Ada jeda)

Sebelum program ini sampai ke pembeli developer kami sudah melakukan testing

jadi sebelum nego kami sudah melakukan berbagai konsekuensi jika nego tidak sesuai ? 
Bukan maksud kami menipu itu karena harga yang sudah di kalkulasi + bantuan tiba tiba di potong akhirnya bantuan / software kadang tidak lengkap


<!-- END LICENSE --> */
#include <algorithm>
#include <cstdint>

#include "argmax.cuh"
#include "common.cuh"
#include "sum.cuh"

static __global__ void argmax_f32(const float * __restrict__ x, int32_t * __restrict__ dst, const int64_t ncols) {
    const int64_t row = blockIdx.x;

    float maxval = -FLT_MAX;
    int   argmax = -1;
    const float * rowx = x + row * ncols;

    for (int32_t col = threadIdx.x; col < ncols; col += blockDim.x) {
        const float val = rowx[col];
        if (val > maxval) {
            maxval = val;
            argmax = col;
        }
    }

#pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1) {
        const float val = __shfl_xor_sync(0xFFFFFFFF, maxval, offset, WARP_SIZE);
        const int   col = __shfl_xor_sync(0xFFFFFFFF, argmax, offset, WARP_SIZE);
        if (val > maxval) {
            maxval = val;
            argmax = col;
        }
    }

    const int n_warps = blockDim.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;
    const int warp_id = threadIdx.x / WARP_SIZE;
    if (n_warps > 1) {
        constexpr int    max_warps = 1024 / WARP_SIZE;
        __shared__ float shared_maxval[max_warps];
        __shared__ int   shared_argmax[max_warps];
        if (lane_id == 0) {
            shared_maxval[warp_id] = maxval;
            shared_argmax[warp_id] = argmax;
        }

        __syncthreads();

        if (warp_id == 0) {
            if (lane_id < n_warps) {
                maxval = shared_maxval[lane_id];
                argmax = shared_argmax[lane_id];
            }
#pragma unroll
            for (int offset = 16; offset > 0; offset >>= 1) {
                const float val = __shfl_xor_sync(0xFFFFFFFF, maxval, offset, WARP_SIZE);
                const int   col = __shfl_xor_sync(0xFFFFFFFF, argmax, offset, WARP_SIZE);
                if (val > maxval) {
                    maxval = val;
                    argmax = col;
                }
            }
        }
    }

    if (warp_id == 0 && lane_id == 0) {
        dst[row] = argmax;
    }
}

void ggml_cuda_argmax(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_I32);

    GGML_ASSERT(ggml_is_contiguous(src0));

    const int64_t ne00  = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    const float * src0_d = (const float *) src0->data;
    int32_t     * dst_d  = (int32_t     *) dst->data;

    hipStream_t stream = ctx.stream();

    const int64_t num_blocks = nrows;
    const int64_t num_threads = std::min<int64_t>(1024, (ne00 + WARP_SIZE - 1) / WARP_SIZE * WARP_SIZE);
    const dim3 blocks_dim(num_threads, 1, 1);
    const dim3 blocks_num(num_blocks, 1, 1);

    argmax_f32<<<blocks_num, blocks_dim, 0, stream>>>(src0_d, dst_d, ne00);
}

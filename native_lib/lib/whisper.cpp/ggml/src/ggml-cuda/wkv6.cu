#include "hip/hip_runtime.h"
/* <!-- START LICENSE -->


This Software / Program / Source Code Created By Developer From Company GLOBAL CORPORATION
Social Media:

   - Youtube: https://youtube.com/@Global_Corporation 
   - Github: https://github.com/globalcorporation
   - TELEGRAM: https://t.me/GLOBAL_CORP_ORG_BOT

All code script in here created 100% original without copy / steal from other code if we copy we add description source at from top code

If you wan't edit you must add credit me (don't change)

If this Software / Program / Source Code has you

Jika Program ini milik anda dari hasil beli jasa developer di (Global Corporation / apapun itu dari turunan itu jika ada kesalahan / bug / ingin update segera lapor ke sub)

Misal anda beli Beli source code di Slebew CORPORATION anda lapor dahulu di slebew jangan lapor di GLOBAL CORPORATION!

Jika ada kendala program ini (Pastikan sebelum deal project tidak ada negosiasi harga)
Karena jika ada negosiasi harga kemungkinan

1. Software Ada yang di kurangin
2. Informasi tidak lengkap
3. Bantuan Tidak Bisa remote / full time (Ada jeda)

Sebelum program ini sampai ke pembeli developer kami sudah melakukan testing

jadi sebelum nego kami sudah melakukan berbagai konsekuensi jika nego tidak sesuai ? 
Bukan maksud kami menipu itu karena harga yang sudah di kalkulasi + bantuan tiba tiba di potong akhirnya bantuan / software kadang tidak lengkap


<!-- END LICENSE --> */
#include "common.cuh"
#include "wkv6.cuh"

static __global__ void rwkv_wkv_f32(const int B, const int T, const int C, const int H, const float * k, const float * v, const float * r, const float * tf, const float * td, const float * s, float * dst) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    const int head_size = CUDA_WKV_BLOCK_SIZE;
    const int batch_i = bid / H;
    const int head_i = bid % H;
    const int state_size = C * head_size;
    const int n_seq_tokens = T / B;

    float state[head_size];
    __shared__ float _k[head_size], _r[head_size], _tf[head_size], _td[head_size];

    #pragma unroll
    for (int i = 0; i < head_size; i++) {
        state[i] = s[batch_i * state_size + head_i * head_size * head_size + i * head_size + tid];
    }

    __syncthreads();
    _tf[tid] = tf[head_i * head_size + tid];
    __syncthreads();

    for (int t = batch_i * n_seq_tokens * C + head_i * head_size + tid; t < (batch_i + 1) * n_seq_tokens * C + head_i * head_size + tid; t += C) {
        __syncthreads();
        _k[tid] = k[t];
        _r[tid] = r[t];
        _td[tid] = td[t];
        __syncthreads();

        const float _v = v[t];
        float y = 0;
        for (int j = 0; j < head_size; j += 4) {
            const float4& k = (float4&)(_k[j]);
            const float4& r = (float4&)(_r[j]);
            const float4& tf = (float4&)(_tf[j]);
            const float4& td = (float4&)(_td[j]);
            float4& s = (float4&)(state[j]);
            float4 kv;

            kv.x = k.x * _v;
            kv.y = k.y * _v;
            kv.z = k.z * _v;
            kv.w = k.w * _v;

            y += r.x * (tf.x * kv.x + s.x);
            y += r.y * (tf.y * kv.y + s.y);
            y += r.z * (tf.z * kv.z + s.z);
            y += r.w * (tf.w * kv.w + s.w);

            s.x = s.x * td.x + kv.x;
            s.y = s.y * td.y + kv.y;
            s.z = s.z * td.z + kv.z;
            s.w = s.w * td.w + kv.w;
        }
        dst[t] = y;
    }

    #pragma unroll
    for (int i = 0; i < head_size; i++) {
        dst[T * C + batch_i * state_size + head_i * head_size * head_size + i * head_size + tid] = state[i];
    }
}

void ggml_cuda_op_rwkv_wkv6(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const float * k_d  = (const float *)dst->src[0]->data;
    const float * v_d  = (const float *)dst->src[1]->data;
    const float * r_d  = (const float *)dst->src[2]->data;
    const float * tf_d = (const float *)dst->src[3]->data;
    const float * td_d = (const float *)dst->src[4]->data;
    const float * s_d  = (const float *)dst->src[5]->data;

    const int64_t B = dst->src[5]->ne[1];
    const int64_t T = dst->src[0]->ne[2];
    const int64_t C = dst->ne[0];
    const int64_t H = dst->src[0]->ne[1];

    float * dst_d = (float *)dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(dst->src[5]->type == GGML_TYPE_F32);
    GGML_ASSERT(C % H == 0);
    GGML_ASSERT(C / H == CUDA_WKV_BLOCK_SIZE); // The current cuda kernel is designed for RWKV6, HEAD_SIZE == 64

    rwkv_wkv_f32<<<B * H, C / H, 0, stream>>>(B, T, C, H, k_d, v_d, r_d, tf_d, td_d, s_d, dst_d);
}
